//cuda inclusion
#include "hip/hip_runtime.h"
#include ""

//Math inclusion
#define _USE_MATH_DEFINES
#include <math.h>
#include <cmath>
#include <ctime>

//c++ and project inclusion
#include <stdio.h>
#include "galaxyKernel.h"
