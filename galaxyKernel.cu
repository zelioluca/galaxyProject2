#include "hip/hip_runtime.h"
//cuda inclusion
#include "hip/hip_runtime.h"
#include ""

//Math inclusion
#define _USE_MATH_DEFINES
#include <math.h>
#include <cmath>
#include <ctime>

//c++ and project inclusion
#include <stdio.h>
#include "galaxyKernel.h"

//define thread x and thread y
constexpr auto TX = 32; 
constexpr auto TY = 32; 
constexpr auto GRID = 100000; 
constexpr auto WINDOW = 1024; 

//Cuda error handling start here
inline void error_check(hipError_t err, const char* file, int line)
{
	if (err != hipSuccess) {
		::fprintf(stderr, "\nCUDA ERROR at %s[%d] : %s\n", file, line, hipGetErrorString(err));
		printf("\nGeneral error at %s[%d] : %s\n", file, line, hipGetErrorString(err));
	}
}

#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)

//this function clamp the numbers ORI with float 
__device__ float ClampValue(float temp, float a, float b)
{
	return fmaxf(a, fminf(b, temp));
}

//this is the kernel
__global__
void LaunchTheGalaxy(float * deviceReal_ascension, float* deviceReal_declination, float * deviceFlat_ascension, float* deviceFlat_declination, unsigned long long int * deviceDD, unsigned long long int * deviceDR, unsigned long long int * deviceRR, size_t size)
{
	__shared__ float temp_real_ascension[1024]; 
	__shared__ float temp_real_declination[1024]; 
	__shared__ float temp_flat_ascension[1024]; 
	__shared__ float temp_flat_declination[1024]; 

	__shared__ float tempDD; 

	int tidX = threadIdx.x + blockIdx.x * blockDim.x;

	if ((threadIdx.x >= size) || (threadIdx.y >= size))
	{
		return; 
	}

	for (int i=tidX; i < WINDOW; i++)
	{
		if((i * WINDOW + tidX) <= GRID)
			temp_real_ascension[i] = deviceReal_ascension[i * WINDOW + tidX];
			temp_real_declination[i] = deviceReal_declination[i * WINDOW + tidX];


		//__syncthreads();

		//for (int row = 0; row < WINDOW; row += blockDim.y)
		//{
		//	for (int col = 0; col < WINDOW; col += blockDim.x)
		//	{
		//		tempDD = (acosf(ClampValue(sinf(temp_real_declination[threadIdx.x + col]) * sinf(temp_real_declination[threadIdx.y + row]) + cosf(temp_real_declination[threadIdx.x + col]) * cosf(temp_real_declination[threadIdx.y + row]) * cos(temp_real_ascension[threadIdx.x + col] - temp_real_ascension[threadIdx.y + row]), -1.0f, 1.0f)) * 720.0f / (float)M_PI);
		//		atomicAdd(deviceDD + (int)tempDD, 1);
		//	}
		//}
		//
	}
	__syncthreads();
}

//this is the kernel handler 
void KernelHandler(float * hostReal_ascension, float * hostReal_declination ,float * hostFlat_ascension, float * hostFlat_declination ,unsigned long long int * hostDD, unsigned long long int* hostDR, unsigned long long int* hostRR, size_t size, size_t histogram)
{
	//allocate memory 
	float * deviceReal_ascension = nullptr;
	float* deviceReal_declination = nullptr;
	float * deviceFlat_ascension = nullptr;
	float* deviceFlat_declination = nullptr;

	unsigned long long int* deviceDD = nullptr; 
	unsigned long long int* deviceDR = nullptr;
	unsigned long long int* deviceRR = nullptr;

	CUDA_CHECK(hipMalloc(&deviceReal_ascension, size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&deviceReal_declination, size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&deviceFlat_ascension, size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&deviceFlat_declination, size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&deviceDD, size * sizeof(unsigned long long int)));
	CUDA_CHECK(hipMalloc(&deviceDR, size * sizeof(unsigned long long int)));
	CUDA_CHECK(hipMalloc(&deviceRR, size * sizeof(unsigned long long int)));

	//copy the array 
	CUDA_CHECK(hipMemcpy(deviceReal_ascension, hostReal_ascension, size * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceReal_declination, hostReal_declination, size * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceFlat_ascension, hostFlat_ascension, size * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceFlat_declination, hostFlat_declination, size * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceDD, hostDD, histogram * sizeof(unsigned long long int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceDR, hostDR, histogram * sizeof(unsigned long long int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceRR, hostRR, histogram * sizeof(unsigned long long int), hipMemcpyHostToDevice));

	//grid and block
	const dim3 blockSize(TX, TY);
	const int bx = ceil((size + blockSize.x - 1) / blockSize.x); 
	const int by = ceil((size + blockSize.y - 1) / blockSize.y);
	const dim3 gridSize(bx, by);

	//print the information on the screen 
	printf("\nSummary: TX is %d TY is %d\n", TX, TY); 

	//start timer
	clock_t kernelStart = clock();

	LaunchTheGalaxy <<<gridSize, blockSize >>> (deviceReal_ascension, deviceReal_declination, deviceFlat_ascension, deviceFlat_declination ,deviceDD, deviceDR, deviceRR, size);

	//check error in the kernel 
	hipError_t errAsync = hipDeviceSynchronize(); 
	hipError_t errSync = hipGetLastError(); 

	if (errSync != hipSuccess)
	{
		printf("\nError in cuda kernel (sync side) %s\n", hipGetErrorString(errSync));

	}
	//check error async
	if (errAsync != hipSuccess)
	{
		printf("\nError in cuda kernel (async side) %s\n", hipGetErrorString(errAsync));
	}

	//end timer 
	clock_t kernelEnd = clock();
	float elapsed = float(kernelEnd - kernelStart) / CLOCKS_PER_SEC;
	printf("\n\nThe kernel timer is %.2f\n\n", elapsed); 

	CUDA_CHECK(hipMemcpy(hostDD, deviceDD, 720 * sizeof(unsigned long long int), hipMemcpyDeviceToHost)); 
	CUDA_CHECK(hipMemcpy(hostDR, deviceDR, 720 * sizeof(unsigned long long int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(hostRR, deviceRR, 720 * sizeof(unsigned long long int), hipMemcpyDeviceToHost));

	//free the cuda memory 
	CUDA_CHECK(hipFree(deviceReal_ascension));
	CUDA_CHECK(hipFree(deviceReal_declination));
	CUDA_CHECK(hipFree(deviceFlat_ascension));
	CUDA_CHECK(hipFree(deviceFlat_declination));
	CUDA_CHECK(hipFree(deviceDD));
	CUDA_CHECK(hipFree(deviceDR));
	CUDA_CHECK(hipFree(deviceRR));
}